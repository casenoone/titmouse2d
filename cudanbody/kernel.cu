#include "hip/hip_runtime.h"
////#include "hip/hip_runtime_api.h"
//#include "math_functions.h"
//#include "hip/hip_math_constants.h"
//#include "hip/hip_runtime.h"
//#include ""
//#include <GL/glut.h>
//#include <stdio.h>
//
//
//
//const float SCREEN_SIZE = 400;
//const float DRAW_SIZE = SCREEN_SIZE / 200 * 10;
//static void key(unsigned char key, int x, int y)
//{
//	switch (key)
//	{
//	case 27: //��ESC
//	case 'q': //��q�����򶼻��˳�
//		exit(0);
//		break;
//	}
//
//	glutPostRedisplay();  //�����ػ�ص�
//}
//
//void drawPoint(double x, double y)
//{
//	//�ں󻺴����ͼ�Σ���һ����
//	glPointSize(7.5f);//ȱʡ��1
//	glBegin(GL_POINTS);
//	glColor3f(1, 128.0 / 255, 51.0 / 255);
//	glVertex3f((x - 1) * DRAW_SIZE, (y - 1) * DRAW_SIZE, 0);
//	glEnd();
//}
//
//void drawLine(double x1, double y1, double x2, double y2) {
//
//	glLineWidth(1);//�����߶ο��
//	glBegin(GL_LINES);
//	glColor3f(1.0, 0.0, 0.0);
//	glVertex2f((x1 - 1) * DRAW_SIZE, (y1 - 1) * DRAW_SIZE); //�������귶Χ
//	glVertex2f((x2 - 1) * DRAW_SIZE, (y2 - 1) * DRAW_SIZE);
//	glEnd();
//	glFlush();
//}
//
//
//static void display(void)
//{
//
//
//	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//	glLoadIdentity();
//	gluLookAt(0, 0, 100, 0, 0, 0, 0, 1, 0);
//
//	drawPoint(1, 1);
//
//	glutSwapBuffers();
//}
//
//static void idle(void)
//{
//
//	glutPostRedisplay();
//
//}
//
//static void resize(int width, int height)
//{
//	const float ar = (float)width / (float)height;
//	glViewport(0, 0, width, height);
//	glMatrixMode(GL_PROJECTION);
//	glLoadIdentity();
//	double ratio = 0.1;
//	glOrtho(-width * 0.5 * ratio, width * 0.5 * ratio, -height * 0.5 * ratio, height * 0.5 * ratio, 2.0, 100.0); //����ʹ������ͶӰ
//	glMatrixMode(GL_MODELVIEW);
//	glLoadIdentity();
//}
//
//
//
//
///*********************����ΪCUDAר��*********************/
//
//__global__ void kernel() {
//
//}
//
//
///*********************����ΪCUDAר��*********************/
//int main(int argc, char** argv)
//{
//
//	/*********************����ΪOpenGL����*********************/
//	glutInit(&argc, argv);
//	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
//	glutInitWindowSize(SCREEN_SIZE, SCREEN_SIZE);
//	glutInitWindowPosition(0, 0);
//	glutCreateWindow("n-body");
//
//	glClearColor(6 / 255.0, 133 / 255.0, 135 / 255.0, 1);
//	glShadeModel(GL_FLAT);
//	/*********************����ΪOpenGL����*********************/
//
//
//
//
//
//
//
//
//
//	return 0;
//
//}

#include <GL/glut.h>
#include <math.h>
#include <string.h>

// ���ڴ�С
const int WINDOW_WIDTH = 800;
const int WINDOW_HEIGHT = 600;

// �ַ���
const char* text = "�����Ѷ����漸�Ѷ�";

// ��Ⱦ�ַ���
void renderText() {
	// �ַ�������
	int len = strlen(text);

	// ��������
	void* font = GLUT_BITMAP_HELVETICA_18;

	// ���ַ�����ȾΪλͼ
	glRasterPos2f(-0.8, 0.0);
	for (int i = 0; i < len; i++) {
		if (text[i] == '��') {
			// ���浱ǰ����
			glPushMatrix();

			// �á���������������ת
			glTranslatef(0.0, -0.02, 0.0);
			glRotatef(45.0 * sin(glutGet(GLUT_ELAPSED_TIME) / 1000.0), 0.0, 0.0, 1.0);
			glTranslatef(0.0, 0.02, 0.0);

			// ��Ⱦ��������
			glutBitmapCharacter(font, text[i]);

			// �ָ�֮ǰ�ľ���
			glPopMatrix();
		}
		else {
			// ��Ⱦ�����ַ�
			glutBitmapCharacter(font, text[i]);
		}
	}
}

// ��ʼ������
void onInit() {
	// ���ô��ڱ���ɫΪ��ɫ
	glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
}

// ���ƺ���
void onDraw() {
	// �����ɫ������
	glClear(GL_COLOR_BUFFER_BIT);

	// �����ַ���
	renderText();

	// ����ǰ�󻺳���
	glutSwapBuffers();
}

int main(int argc, char* argv[]) {
	// ��ʼ��OpenGL��GLUT��
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(WINDOW_WIDTH, WINDOW_HEIGHT);
	glutCreateWindow("OpenGL Window");
	// ע���ʼ�������ͻ��ƺ���
	glutDisplayFunc(onDraw);
	glutIdleFunc(onDraw);
	onInit();
	// ������ѭ��
	glutMainLoop();
	return 0;
}
