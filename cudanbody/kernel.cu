#include "hip/hip_runtime.h"
//#include "hip/hip_runtime_api.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h"
#include "hip/hip_runtime.h"
#include ""
#include <GL/glut.h>
#include <stdio.h>



const float SCREEN_SIZE = 400;
const float DRAW_SIZE = SCREEN_SIZE / 200 * 10;
static void key(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 27: //��ESC
	case 'q': //��q�����򶼻��˳�
		exit(0);
		break;
	}

	glutPostRedisplay();  //�����ػ�ص�
}

void drawPoint(double x, double y)
{
	//�ں󻺴����ͼ�Σ���һ����
	glPointSize(7.5f);//ȱʡ��1
	glBegin(GL_POINTS);
	glColor3f(1, 128.0 / 255, 51.0 / 255);
	glVertex3f((x - 1) * DRAW_SIZE, (y - 1) * DRAW_SIZE, 0);
	glEnd();
}

void drawLine(double x1, double y1, double x2, double y2) {

	glLineWidth(1);//�����߶ο��
	glBegin(GL_LINES);
	glColor3f(1.0, 0.0, 0.0);
	glVertex2f((x1 - 1) * DRAW_SIZE, (y1 - 1) * DRAW_SIZE); //�������귶Χ
	glVertex2f((x2 - 1) * DRAW_SIZE, (y2 - 1) * DRAW_SIZE);
	glEnd();
	glFlush();
}


static void display(void)
{


	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();
	gluLookAt(0, 0, 100, 0, 0, 0, 0, 1, 0);

	drawPoint(1, 1);

	glutSwapBuffers();
}

static void idle(void)
{

	glutPostRedisplay();

}

static void resize(int width, int height)
{
	const float ar = (float)width / (float)height;
	glViewport(0, 0, width, height);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	double ratio = 0.1;
	glOrtho(-width * 0.5 * ratio, width * 0.5 * ratio, -height * 0.5 * ratio, height * 0.5 * ratio, 2.0, 100.0); //����ʹ������ͶӰ
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
}




/*********************����ΪCUDAר��*********************/

__device__ float getSofteningSquared() {
	return 0.001f;
}


__device__
float3 bodyPairInteraction(float4 pi, float4 pj, float3 ai)
{
	float3 distVector = { 0.0f, 0.0f, 0.0f };
	distVector.x = pj.x - pi.x;
	distVector.y = pj.y - pi.y;
	distVector.z = pj.z - pi.z;

	float distSquared = distVector.x * distVector.x + distVector.y * distVector.y
		+ distVector.z * distVector.z;
	distSquared += getSofteningSquared();
	float invDist = rsqrtf(distSquared);
	float invDistCubic = invDist * invDist * invDist;
	float coff = pj.w * invDistCubic;

	ai.x += distVector.x * coff;
	ai.y += distVector.y * coff;
	ai.z += distVector.z * coff;
	return ai;
}

__device__
float3 calcGravitationForce(float4 bodyPos, float4* positions, int numTiles, thread_block cta)
{
	extern __shared__ float4 sharedPos[];

	float3 acc = { 0.0f, 0.0f, 0.0f };

	for (unsigned int tile = 0; tile < numTiles; ++tile)
	{
		sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];
		sync(cta);

		// tile calculation.
#pragma unroll 128
		for (unsigned int index = 0; index < blockDim.x; ++index)
		{
			acc = bodyPairInteraction(bodyPos, sharedPos[index], acc);
		}
		sync(cta);
	}

	return acc;
}


/*********************����ΪCUDAר��*********************/
int main(int argc, char** argv)
{


	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	glutInitWindowSize(SCREEN_SIZE, SCREEN_SIZE);
	glutInitWindowPosition(0, 0);
	glutCreateWindow("n-body");

	glClearColor(6 / 255.0, 133 / 255.0, 135 / 255.0, 1);
	glShadeModel(GL_FLAT);



	return 0;

}
