
#include <hip/hip_runtime.h>
//#include <iostream>
//#include <fstream>
//#include <cmath>
//#include <string>
//
//#include "../../titmouse2d/titmouse2d/src/random.h"
//#include <GL/glut.h>
//#include <cmath>
//#include <cstdio>
//#include <cuda_runtime.h>
//#include "helper_cuda.h"
//
//const float SCREEN_SIZE = 400;
//const float DRAW_SIZE = SCREEN_SIZE / 200 * 10;
//static void key(unsigned char key, int x, int y)
//{
//	switch (key)
//	{
//	case 27: //��ESC
//	case 'q': //��q�����򶼻��˳�
//		exit(0);
//		break;
//	}
//
//	glutPostRedisplay();  //�����ػ�ص�
//}
//
//void drawPoint(double x, double y)
//{
//	//�ں󻺴����ͼ�Σ���һ����
//	glPointSize(2.5f);//ȱʡ��1
//	glBegin(GL_POINTS);
//	glColor3f(1, 128.0 / 255, 51.0 / 255);
//	glVertex3f((x - 1) * DRAW_SIZE, (y - 1) * DRAW_SIZE, 0);
//	glEnd();
//}
//
//void drawLine(double x1, double y1, double x2, double y2) {
//
//	glLineWidth(1);//�����߶ο��
//	glBegin(GL_LINES);
//	glColor3f(1.0, 0.0, 0.0);
//	glVertex2f((x1 - 1) * DRAW_SIZE, (y1 - 1) * DRAW_SIZE); //�������귶Χ
//	glVertex2f((x2 - 1) * DRAW_SIZE, (y2 - 1) * DRAW_SIZE);
//	glEnd();
//	glFlush();
//}
//
//
//
//
//static void idle(void)
//{
//
//	glutPostRedisplay();
//
//}
//
//static void resize(int width, int height)
//{
//	const float ar = (float)width / (float)height;
//	glViewport(0, 0, width, height);
//	glMatrixMode(GL_PROJECTION);
//	glLoadIdentity();
//	double ratio = 0.1;
//	glOrtho(-width * 0.5 * ratio, width * 0.5 * ratio, -height * 0.5 * ratio, height * 0.5 * ratio, 2.0, 100.0); //����ʹ������ͶӰ
//	glMatrixMode(GL_MODELVIEW);
//	glLoadIdentity();
//}
//
//
//
///********************����ΪCUDA����********************/
////__global__ void kernel() {
////	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
////	unsigned int tnum = blockDim.x * gridDim.x;
////	printf("Thread %d %d\n", tid, tnum);
////
////}
//
//
////ͳһ�ڴ漼��
//__global__ void kernel(int* arr, int n) {
//	for (int i = 0; i < n; ++i) {
//		arr[i] = i;
//	}
//}
//
//
///********************����ΪCUDA����********************/
//
//
//
//
//
//
//static void display(void) {
//	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//	glLoadIdentity();
//	gluLookAt(0, 0, 100, 0, 0, 0, 0, 1, 0);
//
//	/**********************����Ϊ��ͼ��**********************/
//
//	drawPoint(1, 1);
//	/**********************����Ϊ��ͼ��**********************/
//
//
//	glutSwapBuffers();
//
//}
//
//int main(int argc, char** argv)
//{
//
//	glutInit(&argc, argv);
//	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
//	glutInitWindowSize(SCREEN_SIZE, SCREEN_SIZE);
//	glutInitWindowPosition(0, 0);
//	glutCreateWindow("titmouse2d");
//	glClearColor(6 / 255.0, 133 / 255.0, 135 / 255.0, 1);
//	glShadeModel(GL_FLAT);
//
//
//
//
//	/**********************����Ϊ��������**********************/
//
//	int n = 32;
//	int* arr;
//	checkCudaErrors(cudaMallocManaged(&arr, n * sizeof(int)));
//
//	kernel << <1, 1 >> > (arr, n);
//	checkCudaErrors(cudaDeviceSynchronize());
//
//	for (int i = 0; i < n; ++i) {
//		printf("%d", arr[i]);
//	}
//	cudaFree(arr);
//
//	/**********************����Ϊ��������**********************/
//
//
//
//
//
//	glutKeyboardFunc(key);       //���̰���ȥʱ
//	glutIdleFunc(idle);          //����ʱ
//	glutReshapeFunc(resize);     //�ı䴰�ڴ�Сʱ
//	glutDisplayFunc(display);    //���ƴ�����ʾʱ
//
//	glutMainLoop();
//
//
//
//
//
//
//	return 0;
//}
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
