
#include <hip/hip_runtime.h>
//#include <iostream>
//#include <fstream>
//#include <cmath>
//#include <string>
//
//#include "../../titmouse2d/titmouse2d/src/random.h"
//#include <GL/glut.h>
//#include <cmath>
//#include <cstdio>
//#include <cuda_runtime.h>
//#include "helper_cuda.h"
//
//const float SCREEN_SIZE = 400;
//const float DRAW_SIZE = SCREEN_SIZE / 200 * 10;
//static void key(unsigned char key, int x, int y)
//{
//	switch (key)
//	{
//	case 27: //按ESC
//	case 'q': //按q键程序都会退出
//		exit(0);
//		break;
//	}
//
//	glutPostRedisplay();  //产生重绘回调
//}
//
//void drawPoint(double x, double y)
//{
//	//在后缓存绘制图形，就一个点
//	glPointSize(2.5f);//缺省是1
//	glBegin(GL_POINTS);
//	glColor3f(1, 128.0 / 255, 51.0 / 255);
//	glVertex3f((x - 1) * DRAW_SIZE, (y - 1) * DRAW_SIZE, 0);
//	glEnd();
//}
//
//void drawLine(double x1, double y1, double x2, double y2) {
//
//	glLineWidth(1);//设置线段宽度
//	glBegin(GL_LINES);
//	glColor3f(1.0, 0.0, 0.0);
//	glVertex2f((x1 - 1) * DRAW_SIZE, (y1 - 1) * DRAW_SIZE); //定点坐标范围
//	glVertex2f((x2 - 1) * DRAW_SIZE, (y2 - 1) * DRAW_SIZE);
//	glEnd();
//	glFlush();
//}
//
//
//
//
//static void idle(void)
//{
//
//	glutPostRedisplay();
//
//}
//
//static void resize(int width, int height)
//{
//	const float ar = (float)width / (float)height;
//	glViewport(0, 0, width, height);
//	glMatrixMode(GL_PROJECTION);
//	glLoadIdentity();
//	double ratio = 0.1;
//	glOrtho(-width * 0.5 * ratio, width * 0.5 * ratio, -height * 0.5 * ratio, height * 0.5 * ratio, 2.0, 100.0); //这里使用正视投影
//	glMatrixMode(GL_MODELVIEW);
//	glLoadIdentity();
//}
//
//
//
///********************以下为CUDA函数********************/
////__global__ void kernel() {
////	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
////	unsigned int tnum = blockDim.x * gridDim.x;
////	printf("Thread %d %d\n", tid, tnum);
////
////}
//
//
////统一内存技术
//__global__ void kernel(int* arr, int n) {
//	for (int i = 0; i < n; ++i) {
//		arr[i] = i;
//	}
//}
//
//
///********************以上为CUDA函数********************/
//
//
//
//
//
//
//static void display(void) {
//	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//	glLoadIdentity();
//	gluLookAt(0, 0, 100, 0, 0, 0, 0, 1, 0);
//
//	/**********************以下为绘图区**********************/
//
//	drawPoint(1, 1);
//	/**********************以上为绘图区**********************/
//
//
//	glutSwapBuffers();
//
//}
//
//int main(int argc, char** argv)
//{
//
//	glutInit(&argc, argv);
//	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
//	glutInitWindowSize(SCREEN_SIZE, SCREEN_SIZE);
//	glutInitWindowPosition(0, 0);
//	glutCreateWindow("titmouse2d");
//	glClearColor(6 / 255.0, 133 / 255.0, 135 / 255.0, 1);
//	glShadeModel(GL_FLAT);
//
//
//
//
//	/**********************以下为主函数区**********************/
//
//	int n = 32;
//	int* arr;
//	checkCudaErrors(cudaMallocManaged(&arr, n * sizeof(int)));
//
//	kernel << <1, 1 >> > (arr, n);
//	checkCudaErrors(cudaDeviceSynchronize());
//
//	for (int i = 0; i < n; ++i) {
//		printf("%d", arr[i]);
//	}
//	cudaFree(arr);
//
//	/**********************以上为主函数区**********************/
//
//
//
//
//
//	glutKeyboardFunc(key);       //键盘按下去时
//	glutIdleFunc(idle);          //空闲时
//	glutReshapeFunc(resize);     //改变窗口大小时
//	glutDisplayFunc(display);    //绘制窗口显示时
//
//	glutMainLoop();
//
//
//
//
//
//
//	return 0;
//}
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//
