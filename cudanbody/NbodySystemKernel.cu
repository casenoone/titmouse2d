#include "hip/hip_runtime.h"
#ifndef _BODYSYSTEM_H_
#define _BODYSYSTEM_H_

#include <math.h>
#include <iostream>
#include <algorithm>

#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

typedef float real;

__constant__ real softeningSquared;

hipError_t setSofteningSquared(real softeningSq)
{
	return hipMemcpyToSymbol(HIP_SYMBOL(
		softeningSquared),
		&softeningSq,
		sizeof(real), 0,
		hipMemcpyHostToDevice);
}

__device__
real getSofteningSquared()
{
	return softeningSquared;
}

__device__
float3 bodyPairInteraction(float4 pi, float4 pj, float3 ai)
{
	float3 distVector = { 0.0f, 0.0f, 0.0f };
	distVector.x = pj.x - pi.x;
	distVector.y = pj.y - pi.y;
	distVector.z = pj.z - pi.z;

	float distSquared = distVector.x * distVector.x + distVector.y * distVector.y
		+ distVector.z * distVector.z;
	distSquared += getSofteningSquared();
	float invDist = rsqrtf(distSquared);
	float invDistCubic = invDist * invDist * invDist;
	float coff = pj.w * invDistCubic;

	ai.x += distVector.x * coff;
	ai.y += distVector.y * coff;
	ai.z += distVector.z * coff;
	return ai;
}

__device__
float3 calcGravitationForce(float4 bodyPos, float4* positions, int numTiles, thread_block cta)
{
	extern __shared__ float4 sharedPos[];

	float3 acc = { 0.0f, 0.0f, 0.0f };

	for (unsigned int tile = 0; tile < numTiles; ++tile)
	{
		sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];
		sync(cta);

		// tile calculation.
#pragma unroll 128
		for (unsigned int index = 0; index < blockDim.x; ++index)
		{
			acc = bodyPairInteraction(bodyPos, sharedPos[index], acc);
		}
		sync(cta);
	}

	return acc;
}

__global__
void integrateBodies(
	float4* newPos,
	float4* oldPos,
	float4* velocity,
	float deltaTime,
	float damping,
	unsigned int numTiles,
	unsigned int numBodies,
	double gamma)
{
	// get thread block.
	thread_block cta = this_thread_block();
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	float4 pos = oldPos[index];

	// compute acceleration.
	float3 accel = calcGravitationForce(pos, oldPos, numTiles, cta);

	// update velocity.
	float4 vel = velocity[index];
	vel.x += accel.x * deltaTime;
	vel.y += accel.y * deltaTime;
	vel.z += accel.z * deltaTime;
	vel.x *= damping;
	vel.y *= damping;
	vel.z *= damping;

	auto x_1 = (newPos - oldPos).x;
	auto temp1 = 1 - std::pow(-oldPos * oldPos / 2 * .0001);
	vel.x += (temp1 * x_1 * gamma) / 2 * 3.14159 * x_1;

	auto y_1 = (newPos - oldPos).y;
	auto temp1 = 1 - std::pow(-oldPos * oldPos / 2 * .0001);
	vel.y += (temp1 * y_1 * gamma) / 2 * 3.14159 * y_1;

	auto z_1 = (newPos - oldPos).z;
	auto temp1 = 1 - std::pow(-oldPos * oldPos / 2 * .0001);
	vel.z += (temp1 * z_1 * gamma) / 2 * 3.14159 * z_1;

	// update position.
	pos.x += vel.x * deltaTime;
	pos.y += vel.y * deltaTime;
	pos.z += vel.z * deltaTime;

	newPos[index] = pos;
	velocity[index] = vel;
}

void integrateNBodySystem(
	real** devicePosition,
	real* deviceVelocity,
	hipGraphicsResource** pGRes,
	volatile unsigned int currentRead,
	float deltaTime,
	float damping,
	unsigned int numBodies,
	int blockSize)
{
	// map buffers.
	cudaGraphicsResourceSetMapFlags(pGRes[currentRead],
		cudaGraphicsMapFlagsReadOnly);
	cudaGraphicsResourceSetMapFlags(pGRes[1 - currentRead],
		cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsMapResources(2, &pGRes[0], 0);
	size_t bytes;
	hipError_t err = hipGetLastError();
	err = hipGraphicsResourceGetMappedPointer((void**)&devicePosition[currentRead],
		&bytes, pGRes[currentRead]);
	err = hipGraphicsResourceGetMappedPointer((void**)&devicePosition[1 - currentRead],
		&bytes, pGRes[1 - currentRead]);

	// launch kernel.
	int numBlocks = (numBodies + blockSize - 1) / blockSize;
	int sharedMemSize = blockSize * 4 * sizeof(real);
	integrateBodies << <numBlocks, blockSize, sharedMemSize >> > (
		(float4*)devicePosition[1 - currentRead],
		(float4*)devicePosition[currentRead],
		(float4*)deviceVelocity,
		deltaTime, damping,
		numBlocks, numBodies);
	hipGraphicsUnmapResources(2, &pGRes[0], 0);
}

#endif